// Vector addition: C = 1/A + 1/B 
// using multiple GPUs with OpenMP
// and the unified memory

// Includes
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <omp.h>          // header for OpenMP
#include <hip/hip_runtime.h>

// Variables
float* h_A;   // host vectors
float* h_B;
float* h_C;
float* h_D;

// Functions
void RandomInit(float*, int);

// Device code
__global__ void VecAdd(const float* A, const float* B, float* C, int N, int NGPU, int cpu_thread_id)
{
    int offset = N/NGPU*cpu_thread_id;
    int i = blockDim.x * blockIdx.x + threadIdx.x + offset;
    if (i < N)
        C[i] = 1.0/A[i] + 1.0/B[i];
    __syncthreads();
}

// Host code

int main(void)
{
    printf("\n");
    printf("Vector Addition with multiple GPUs \n");
    printf("using unified memory \n");
    printf("\n");

    int N, NGPU, cpu_thread_id=0;
    int *Dev; 
//    long mem = 1024*1024*1024;     // 4 Giga for float data type.

    printf("Enter the number of GPUs: ");
    scanf("%d", &NGPU);
    printf("%d\n", NGPU);
    Dev = (int *)malloc(sizeof(int)*NGPU);

    int numDev = 0;
    printf("GPU device number: ");
    for(int i = 0; i < NGPU; i++) {
      scanf("%d", &Dev[i]);
      printf("%d ",Dev[i]);
      numDev++;
      if(getchar() == '\n') break;
    }
    printf("\n");
    if(numDev != NGPU) {
      fprintf(stderr,"Should input %d GPU device numbers\n", NGPU);
      exit(1);
    }

    printf("Enter the size of the vectors: ");
    scanf("%d", &N);        
    printf("%d\n", N);        
//    if (3*N > mem) {
//        printf("The size of these 3 vectors cannot be fitted into 4 Gbyte\n");
//        exit(1);
//    }
    long size = N*sizeof(float);

    // Set the sizes of threads and blocks
    int threadsPerBlock;
    printf("Enter the number of threads per block: ");
    scanf("%d", &threadsPerBlock);
    printf("%d\n", threadsPerBlock);
    if(threadsPerBlock > 1024) {
      printf("The number of threads per block must be less than 1024 ! \n");
      exit(1);
    }
    int blocksPerGrid = (N + threadsPerBlock*NGPU - 1) / (threadsPerBlock*NGPU);
    printf("The number of blocks is %d\n", blocksPerGrid);
    if(blocksPerGrid > 2147483647) {
      printf("The number of blocks must be less than 2147483647 ! \n");
      exit(1);
    }

    // Allocate Unified Memory -- accessible to CPU and GPU

    hipMallocManaged((void**)&h_A, size);
    hipMallocManaged((void**)&h_B, size);
    hipMallocManaged((void**)&h_C, size);

    // Initialize input vectors

    RandomInit(h_A, N);
    RandomInit(h_B, N);

    // timer
    hipEvent_t start, stop;

    float gputime;

    omp_set_num_threads(NGPU);

    #pragma omp parallel private(cpu_thread_id)
    {
	cpu_thread_id = omp_get_thread_num();
	hipSetDevice(Dev[cpu_thread_id]);
//	cudaSetDevice(cpu_thread_id);

        // start the timer
        if(cpu_thread_id == 0) { 
          hipEventCreate(&start);
          hipEventCreate(&stop);
          hipEventRecord(start,0);
        }

        VecAdd<<<blocksPerGrid, threadsPerBlock>>>(h_A, h_B, h_C, N, NGPU, cpu_thread_id);

        // stop the timer

	if(cpu_thread_id == 0) {
          hipEventRecord(stop,0);
          hipEventSynchronize(stop);
          hipEventElapsedTime( &gputime, start, stop);
          printf("Processing time for GPU: %f (ms) \n",gputime);
          printf("GPU Gflops: %f\n",3*N/(1000000.0*gputime));
	}

	hipDeviceSynchronize();

    } 

    // start the timer
    hipEventRecord(start,0);

    h_D = (float*)malloc(size);     // compute the reference solution
    for(int i = 0; i < N; ++i) 
      h_D[i] = 1.0/h_A[i] + 1.0/h_B[i];
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",3*N/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/gputime);

    // Destroy timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result
    printf("Check result:\n");
    double sum=0; 
    double diff;
    for (int i = 0; i < N; ++i) {
        diff = abs(h_D[i] - h_C[i]);
        sum += diff*diff; 
    }
    sum = sqrt(sum);
    printf("norm(h_C - h_D)=%20.15e\n",sum);

    for (int i=0; i < NGPU; i++) {
	hipSetDevice(i);
	hipDeviceReset();
    }

    return 0;
}


// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = rand() / (float)RAND_MAX;
}
