/* This program uses the host CURAND API to generate pseudorandom numbers */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

int main(void)
{
  int i,n;
  float *devData, *hostData;
  hiprandGenerator_t gen;      

  printf("Enter the number of random numbers to be generated: ");
  scanf("%d",&n);
  printf("%d\n",n);
  printf("\n");

  /* Allocate n floats on host */
  hostData = (float*)malloc(n*sizeof(float));

  /* Allocate n floats on device */
  hipMalloc((void **)&devData, n*sizeof(float));

  /* Create pseudo-random number generator */

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

  /* Set seed */
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

  /* Generate n floats on device */
  hiprandGenerateUniform(gen, devData, n);

  /* Copy device memory to host */
  hipMemcpy(hostData, devData, n*sizeof(float), hipMemcpyDeviceToHost);

  /* Show result */
  for(i = 0; i < n; i++) {
    printf("%.5e \n", hostData[i]);
  }
  printf("\n");

  /* Cleanup */
  hiprandDestroyGenerator(gen);
  hipFree(devData);
  free(hostData);

}
