/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. 
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>


/**
 * Get parameters from STDIN.
 */
static void read_from_stdin(int *gid, int *n)
{
    char *s, buf[1024];

    fgets(buf, 1023, stdin);
    if ((s = strchr(buf, '#')) != NULL) *s = '\0';
    *gid = atoi(buf);

    fgets(buf, 1023, stdin);
    if ((s = strchr(buf, '#')) != NULL) *s = '\0';
    *n = atoi(buf);
}

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */

__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int main(int argc, char **argv)
{
    int gid, numElements;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    read_from_stdin(&gid, &numElements);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
	printf("!!! Cannot select GPU with device ID = %d\n", gid);
	exit(1);
    }
    printf("Choose GPU with device ID = %d\n", gid);

    // Print the vector length to be used, and compute its size
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the input vector A in the host (CPU)
    float *h_A = (float *)malloc(size);

    // Allocate the input vector B in the host
    float *h_B = (float *)malloc(size);

    // Allocate the output vector C in the host 
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors with random numbers with uniform distribution in (0,1)
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the input vector A in device 
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the input vector B in device
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the output vector C in device
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the input vectors A and B from the host memory to the device memory

    printf("Copy input vectors from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

