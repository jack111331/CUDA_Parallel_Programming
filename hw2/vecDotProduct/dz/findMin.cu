// Vector Dot Product A.B
// compile with the following command:
//
//
// (for GTX1060)
// nvcc -arch=compute_61 -code=sm_61,sm_61 -O3 -m64 -o vecAdd vecAdd.cu


// Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Variables
float* h_A;   // host vectors
float* h_C;
float* d_A;   // device vectors
float* d_C;

// Functions
void RandomInit(float*, int);

// Device code
__global__ void minOfArray(const float* A, float* C, int N)
{
    extern __shared__ float cache[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int cacheIndex = threadIdx.x;

    float temp = 2.0;  // register for each thread
    while (i < N) {
    	if (A[i] < temp) {
	        temp = A[i];
    	}
        i += blockDim.x*gridDim.x;  
    }
   
    cache[cacheIndex] = temp;   // set the cache value 

    __syncthreads();

    // perform parallel reduction, threadsPerBlock must be 2^m

    int ib = blockDim.x/2;
    while (ib != 0) {
      if(cacheIndex < ib) {
      	if (cache[cacheIndex + ib] < cache[cacheIndex]) {
	      cache[cacheIndex] = cache[cacheIndex + ib]; 
	    }      
      }
      __syncthreads();

      ib /=2;
    }
    
    if(cacheIndex == 0)
      C[blockIdx.x] = cache[0];

}

// Host code

int main(void)
{

    int gid;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    printf("Enter the GPU ID: ");
    scanf("%d",&gid);
    printf("%d\n", gid);
    err = hipSetDevice(gid);
    if (err != hipSuccess) {
        printf("!!! Cannot select GPU with device ID = %d\n", gid);
        exit(1);
    }
    printf("Set GPU with device ID = %d\n", gid);

    hipSetDevice(gid);

    printf("Minimum of array of A\n");
    int N;

    printf("Enter the size of the vectors: ");
    scanf("%d",&N);        
    printf("%d\n",N);        

    // Set the sizes of threads and blocks

    int threadsPerBlock;
    printf("Enter the number (2^m) of threads per block: ");
    scanf("%d",&threadsPerBlock);
    printf("%d\n",threadsPerBlock);
    if( threadsPerBlock > 1024 ) {
      printf("The number of threads per block must be less than 1024 ! \n");
      exit(0);
    }

//    int blocksPerGrid = (N + threadsPerBlock - 1)/threadsPerBlock;
//    printf("The number of blocks per grid:%d\n",blocksPerGrid);
 
    int blocksPerGrid;
    printf("Enter the number of blocks per grid: ");
    scanf("%d",&blocksPerGrid);
    printf("%d\n",blocksPerGrid);

    if( blocksPerGrid > 2147483647 ) {
      printf("The number of blocks must be less than 2147483647 ! \n");
      exit(0);
    }

    // Allocate input vectors h_A and h_B in host memory

    int size = N * sizeof(float);
    int sb = blocksPerGrid * sizeof(float);

    h_A = (float*)malloc(size);
    h_C = (float*)malloc(sb);     // contains the result of dot-product from each block
    
    // Initialize input vectors

    RandomInit(h_A, N);


    // create the timer
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start the timer
    hipEventRecord(start,0);

    // Allocate vectors in device memory

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_C, sb);

    // Copy vectors from host memory to device memory

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Intime;
    hipEventElapsedTime( &Intime, start, stop);
    printf("Input time for GPU: %f (ms) \n",Intime);

    // start the timer
    hipEventRecord(start,0);

    int sm = threadsPerBlock*sizeof(float);
    minOfArray <<< blocksPerGrid, threadsPerBlock, sm >>>(d_A, d_C, N);
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float gputime;
    hipEventElapsedTime( &gputime, start, stop);
    printf("Processing time for GPU: %f (ms) \n",gputime);
    printf("GPU Gflops: %f\n",N/(1000000.0*gputime));
    
    // Copy result from device memory to host memory
    // h_C contains the result of each block in host memory

    // start the timer
    hipEventRecord(start,0);

    hipMemcpy(h_C, d_C, sb, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_C);

    double h_G=2.0;
    for(int i = 0; i < blocksPerGrid; i++) {
      if (h_C[i] < h_G) {
      	h_G = h_C[i];
      }
    }
    

    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float Outime;
    hipEventElapsedTime( &Outime, start, stop);
    printf("Output time for GPU: %f (ms) \n",Outime);

    float gputime_tot;
    gputime_tot = Intime + gputime + Outime;
    printf("Total time for GPU: %f (ms) \n",gputime_tot);

    // start the timer
    hipEventRecord(start,0);

    // to compute the reference solution

    double h_D=2.0;       
    for(int i = 0; i < N; i++) {
      if (h_A[i] < h_D) {
      	h_D = h_A[i];
      }
    }
    
    // stop the timer
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);

    float cputime;
    hipEventElapsedTime( &cputime, start, stop);
    printf("Processing time for CPU: %f (ms) \n",cputime);
    printf("CPU Gflops: %f\n",N/(1000000.0*cputime));
    printf("Speed up of GPU = %f\n", cputime/(gputime_tot));

    // destroy the timer
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // check result

    printf("Check result:\n");
    double diff = abs( (h_D - h_G)/h_D );
    printf("|(h_G - h_D)/h_D|=%20.15e\n",diff);
    printf("h_G =%20.15e\n",h_G);
    printf("h_D =%20.15e\n",h_D);

    free(h_A);
    free(h_C);

    hipDeviceReset();
}


// Allocates an array with random float entries in (-1,1)
void RandomInit(float* data, int n)
{
    for (int i = 0; i < n; ++i)
        data[i] = 2.0*rand()/(float)RAND_MAX - 1.0;
}



